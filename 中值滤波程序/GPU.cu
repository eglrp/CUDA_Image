#include "hip/hip_runtime.h"
//
//  GPU.cpp
//  hpalab5
//
//  Created by Harshdeep Singh Chawla on 10/11/16.
//  Copyright ?2016 Harshdeep Singh Chawla. All rights reserved.
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "MedianFilter.h"
#include "Bitmap.h"
void CUDA_CALL(hipError_t x)
{
	if (x != hipSuccess)
	{
		printf("\nCUDA Error:%s(err_num=%d)\n",hipGetErrorString(x),x);
		hipDeviceReset(); 
		hipSetDevice(0);
		//exit(0);
	}
}
///Kernelk function
__global__ void MFKernel(unsigned char *inputImage, unsigned char *outputImage, int Width, int Height)
{
	// indexing for thread.
	 int idy = blockIdx.y * blockDim.y + threadIdx.y;
	 int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//filter mask
	 char filter[9];
	 if ((idy >= Height) || (idx >= Width))
		 return;
	/////checking boundry conditions
	if((idy==0) || (idx==0) || (idy==Height-1) || (idx==Width-1))
				outputImage[idy*Width+idx] = 0;
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) { 
			for (int y = 0; y < WINDOW_SIZE; y++){
				filter[x*WINDOW_SIZE+y] = inputImage[(idy+y-1)*Width+(idx+x-1)];   // setup the filterign window.
			}
		}
		////Sorting in filter
		for (int i = 0; i < 9; i++) {
			for (int j = i + 1; j < 9; j++) {
				if (filter[i] > filter[j]) { 
					//Swap the variables.
					char tmp = filter[i];
					filter[i] = filter[j];
					filter[j] = tmp;
				}
			}
		}
		outputImage[idy*Width+idx] = filter[4];   //Set output variables.
	}
}
///GPU Function
void MedianFilterGPU(Bitmap* image, Bitmap* outputImage){

	//Cuda error and image values.
	/*hipError_t status;*/
	int w = image->Width();
	int h = image->Height();
	size_t bytes = w * h * sizeof(unsigned char);
	//initialize images.
	hipError_t err;
	unsigned char *inputimage_d;
	err=hipMalloc((void**)&inputimage_d, bytes);
	CUDA_CALL(err);
	err = hipSuccess;
	err=hipMemcpy(inputimage_d, image->image, bytes, hipMemcpyHostToDevice);
	CUDA_CALL(err);
	err = hipSuccess;
	unsigned char *outputImage_d;
	err=hipMalloc((void**)&outputImage_d, bytes);
	CUDA_CALL(err);
	err = hipSuccess;
	//set block and grids.
	int TILE_SIZE = 16;
	dim3 dimBlock(TILE_SIZE, TILE_SIZE,1);
	dim3 dimGrid((w+15)/16,(h+15)/16,1);
	//std::cout << "width=:" << w << "height=:" << h << std::endl;
	MFKernel <<<dimGrid,dimBlock>>>(inputimage_d, outputImage_d, w, h);
	hipDeviceSynchronize();
	err=hipGetLastError();
	CUDA_CALL(err);
	err = hipSuccess;
	// save output image to host.
	err=hipMemcpy(outputImage->image, outputImage_d, bytes, hipMemcpyDeviceToHost);
	CUDA_CALL(err);
	err = hipSuccess;
	//Free the memory
	hipFree(inputimage_d);
	hipFree(outputImage_d);
	return ;
}
